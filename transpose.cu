
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <fstream>
#include <iostream>
#include <omp.h>
//#define DEBUG

using namespace std;
extern double deviceCall_time;
//extern double omptime;

void cudasafe( hipError_t error, char* message)
{
     if(error!=hipSuccess) { fprintf(stderr,"ERROR: %s : %i\n",message,error); exit(-1); }
}

// assume matrix 1 should be transposed to matrix 2.
__global__ void transpose(double *mat1,double *mat2,const int row,const int col)
{
	int idx=threadIdx.x,idy=threadIdx.y;

	int globalBlkIndex=blockIdx.y*blockDim.y*col + blockIdx.x*blockDim.x;       //multiply by column 
	int globalIndexWrite=blockIdx.x*blockDim.x*row + blockIdx.y*blockDim.y;     //multiply by row

	__shared__ double shmat1[16][16];         //Shared mem equal to number of thread
//	__shared__ int shmat2[16][16]; 

	shmat1[idy][idx]=mat1[globalBlkIndex + idy*col+idx];    //Read Tile from mat1 row wise and write to shared memory row wise

	__syncthreads();

	mat2[globalIndexWrite + idy*row+idx]=shmat1[idx][idy];   //Read from Shared memory column wise and write to mat2 row wise

}

// This function recieves the original N*K matrix from the CPU and performs the Clustering on it.Also the values of N(Centroids) and K(data points) are passed into it. The Cpu also takes as input d_output where it will store the matrix in the transposed form. The dimensions of d_output is K*N
double* call_transpose(double *h_input,int centroids,int data_points)
{
	double *d_input,*d_output;
	
	cudasafe(hipMalloc(&d_input,sizeof(double)*centroids*data_points),"Error in allocating memory to d_input");
	cudasafe(hipMalloc(&d_output,sizeof(double)*centroids*data_points),"Error in allocating memory to d_output");

	cudasafe(hipMemcpy(d_input,h_input,sizeof(double)*centroids*data_points,hipMemcpyHostToDevice),"Error in copying data from the h_input to d_input");

	
	dim3 block(16,16);                      //multiple of 16 total 256 threads
	dim3 grid(centroids/16,data_points/16);   //number of blocks //
	
	//trans1<<<grid,block>>>(d_mat1,d_mat2,row,col);
	clock_t begin = clock();
	//omptime=omp_get_wtime();
	transpose<<<grid,block>>>(d_input,d_output,data_points,centroids);

	//cudasafe(cudaPeekAtLastError(),"errors occured at kernel");

	cudasafe(hipDeviceSynchronize(),"Error in Transpose Kernel");
	//omptime=omp_get_wtime()-omptime;
	clock_t end = clock();
	//printf("Transpose time %lf\n",omptime);
	deviceCall_time += ((double)(end - begin)/CLOCKS_PER_SEC);

	//Below portion of the code is to print and check the value of the transposed matrix
       
	#ifdef DEBUG1

		fstream fout;

		double* h_output=(double *)malloc(sizeof(double)*data_points*centroids);  //declare an array in host to hold the transposed matrix values

		cudasafe(cudaMemcpy(h_output,d_output,sizeof(double)*data_points*centroids,cudaMemcpyDeviceToHost),"Copying from d_output to h_output");  //copy from d_output to h_output

		fout.open("outputtranspose.txt",ios::out);
		//printf("\n");
		for(int i=0;i<centroids;i++)
		{
			//printf("\n");
			fout << endl;

			for(int j=0;j<data_points;j++)
			{
				fout << h_output[i*data_points + j ]<< " ";
				//if(h_output[i*data_points + j] <=0 )
				//	printf("%f \n",h_output[i*data_points + j]);
			}
		}	

		fout<<endl;

		fout.close();

		free(h_output);
		

	#endif

	hipFree(d_input);

	return d_output;
	
}
