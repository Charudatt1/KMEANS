
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <omp.h>
#define DEBUG
#define GRID_SIZE 1024

/*To find dimensions of new centroids
step 1: sum all documents belonging to same cluster
step 2: divide sum in step 1 by document count per cluster
*/
using namespace std;
extern double deviceCall_time;
//extern double omptime;
const unsigned long long offset=4294967295;

void cudasafe( hipError_t error, char* message);

template <class T> const T& min (const T& a, const T& b) {
  return !(b<a)?a:b;     // or: return !comp(b,a)?a:b; for version (2)
}

//something


//To do sum of all documents belonging to same cluster
//OPTIMIZED KERNEL//

__global__ void centroid_cal(double *d_data,double *d_centroid,unsigned long long *d_min,int start_offset,int end_offset,int Bdim,int Dim,int size)
{
	
	extern __shared__ unsigned long long sharemin[];					//size of sharemin array is assumed to be equal to blockDim.x

	//unsigned long long offset2=4294967295;
	//int threadID=threadIdx.x;
	int gid=threadIdx.x + blockIdx.x*blockDim.x;						//get the global id

	int factor = (int)ceil((double)size/blockDim.x);	//To decide each thread will read how many elements from d_min 

	int centroid_num;
	int doc_num;
	
	for(int i=0;i<factor;i++)
	{
		int curroffset=start_offset + i*blockDim.x +threadIdx.x; 		//calculate the current offset for copying the values between start_offset to end_offset

		if(curroffset <= end_offset )									//TODO replace this condition with hack branching
			sharemin[ threadIdx.x ]=d_min[curroffset];
		else
			sharemin[ threadIdx.x ]=ULLONG_MAX;							//If no more values exist or we have crossed the end_offset

	//	__syncthreads();												//wait for the shared memory values to be filled


		for(int j=0; j<blockDim.x ;j++)												//Indefinite Loop
		{
			if( sharemin[j] == ULLONG_MAX )  							// No more values exist
				break;

			//count++;

			centroid_num = (int)(sharemin[j] & offset);
			doc_num = (int)((sharemin[j] >>32) & offset);

			if(gid<Dim)																			   //check global limit for dimension
					d_centroid[blockIdx.x*blockDim.x*Bdim + threadIdx.x*Bdim+centroid_num]+=d_data[doc_num*Dim + blockIdx.x*blockDim.x+threadIdx.x]; //Data is stored in d x k format
			
		}

	}

	//if(gid==0)
	//	printf("%d\n",count);

}



double *calculate_new_centroid(double *h_data,double *h_centroid,unsigned long long *h_min,unsigned long long *h_unique,int Adim,int Bdim,int dim)
{
		int i=0;
        double *d_data,*d_centroid;

        double *new_centroids=(double *)malloc(sizeof(double)*Bdim*dim);
        
        unsigned long long *d_min;//*d_unique;

        cudasafe(hipMalloc(&d_data,sizeof(double)*Adim*dim),"Allocating mem in d_data in new centroid");
        cudasafe(hipMalloc(&d_centroid,sizeof(double)*Bdim*dim),"Allocating mem in d_centroid in new centroid");
	    cudasafe(hipMalloc(&d_min,sizeof(unsigned long long)*Adim),"Allocating mem in d_min in new centroid");
	    //cudaMalloc(&d_unique,sizeof(unsigned long long)*Bdim);

		int NUMBEROFSTREAM=min(16,(int)(ceil((double)480/dim)));

        hipStream_t stream[NUMBEROFSTREAM];
        hipMemcpy(d_data,h_data,sizeof(double)*Adim*dim, hipMemcpyHostToDevice);
       // cudaMemcpy(d_unique,h_unique,sizeof(double)*Bdim, cudaMemcpyHostToDevice);
	    hipMemcpy(d_min,h_min,sizeof(unsigned long long)*Adim,hipMemcpyHostToDevice);    
	
		//Create number of streams = number of centroids
	    for (i = 0; i < NUMBEROFSTREAM ;i++) 
	    {
	  	          hipStreamCreate(&stream[i]);  	
		}

		int grid=((dim<=480)? 1 : ((int)ceil((double)dim/1024)));
	    //Number of threads equals to dimension or maximum value 1024
	    int block=min(dim,1024);
	    int size;

	    int len=(int)(ceil((double)Bdim/NUMBEROFSTREAM));
	    clock_t begin = clock();
	    //omptime=omp_get_wtime();
	    for(i=0;i<NUMBEROFSTREAM;i++) 
	    {
	    	if((i*len) >= Bdim )
	        	break;
	        	
	       	int start_offset=h_unique[i*len];
	       	//int last_pos=((((i+1)*len)-1) >=Bdim) ? (Bdim-1) : (h_unique((i+1)*len)-1);       		
	       	int end_offset=(((((i+1)*len)) >=Bdim) ? (Adim-1) : (h_unique[(i+1)*len]-1));       		
	        //int end_offset=h_unique[last_pos];
	        size=end_offset - start_offset + 1;		//size of each array for the shared memory	
           centroid_cal<<<grid,block,block*sizeof(unsigned long long),stream[i]>>>(d_data,d_centroid,d_min,start_offset,end_offset,Bdim,dim,size); //inclusive of start offset and inclusive of end offset
		}
		//omptime=omp_get_wtime()-omptime;
		//printf("New Centroid time %lf\n",omptime);
		clock_t end = clock();
		deviceCall_time += ((double)(end - begin)/CLOCKS_PER_SEC);
	        
        for(i=0;i<NUMBEROFSTREAM;i++) 
        {
        	//Wait till all streams complete their operations
            hipStreamSynchronize(stream[i]);
        }
  	
	    for (i=0; i < NUMBEROFSTREAM; i++) 
        {
	        hipStreamDestroy(stream[i]);
		}

		hipDeviceSynchronize();

		hipMemcpy(new_centroids,d_centroid,sizeof(double)*Bdim*dim, hipMemcpyDeviceToHost);

	    //Free unwanted memory allocations in GPU    
		
		hipFree(d_data);	
	    hipFree(d_min);       
        hipFree(d_centroid);
       // cudaFree(d_unique);
      /*  for(int i=0;i<(Bdim);i++) 
	    {
	    	
	    	for (int j = 0; j < dim; j++)
	    	{
	    		cout<<new_centroids[j*Bdim+i]<<" ";

	    	}
	    	cout<<endl;
	    	      
	    }*/   
        return new_centroids;
}



