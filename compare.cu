
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <omp.h>
//#include "gputimer.h"
#define DEBUG
#define THRESHOLD 1

using namespace std;

extern double deviceCall_time;
void cudasafe( hipError_t error, char* message);
//extern double omptime;
__global__ void comp_cent(double *d_centold,double *d_centnew,double basevalue,int *flag)
{
	__shared__ int *s_flag;
	s_flag=flag;

	double base=basevalue;

	double diff=abs(d_centnew[blockIdx.x*blockDim.x+threadIdx.x]-d_centold[blockIdx.x*blockDim.x+threadIdx.x]);

	if(base < diff)
		*s_flag=0;

}
int compare_centroids(double *h_centroid,double *new_centroid,int Bdim,int dim)
{
	double *d_centold,*d_centnew;
	int tmp=1;
	int *h_flag=&tmp; //(int *)malloc(sizeof(int));
	int *d_flag;
	double thre_shold=THRESHOLD;

	cudasafe(hipMalloc(&d_centold,sizeof(double)*Bdim*dim),"Error compare d_centold");
	cudasafe(hipMalloc(&d_centnew,sizeof(double)*Bdim*dim),"Error compare d_centnew");
	cudasafe(hipMalloc(&d_flag,sizeof(int)),"Error compare d_flag");
	

	hipMemcpy(d_centold,h_centroid,sizeof(double)*Bdim*dim,hipMemcpyHostToDevice);
	hipMemcpy(d_centnew,new_centroid,sizeof(double)*Bdim*dim,hipMemcpyHostToDevice);
	hipMemcpy(d_flag,h_flag,sizeof(int),hipMemcpyHostToDevice);

	int size=Bdim*dim;
	int block=min(1024,size);					//minimum of 1024 or total numer of cells in h_centroid matrix
	int grid=(int)ceil((double)size/block);		

	clock_t begin = clock();
	//omptime=omp_get_wtime();
	comp_cent<<<grid,block>>>(d_centold,d_centnew,thre_shold,d_flag);
	//omptime=omp_get_wtime()-omptime;
	clock_t end = clock();
	//printf("Compare time %lf\n",omptime);
	deviceCall_time += ((double)(end - begin)/CLOCKS_PER_SEC);

	hipDeviceSynchronize();
	hipMemcpy(h_flag,d_flag,sizeof(int),hipMemcpyDeviceToHost);

	//Free Memory

	hipFree(d_centold);
	hipFree(d_centnew);
	hipFree(d_flag);


	return (*h_flag);
}