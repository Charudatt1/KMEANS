
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <fstream>
#include <iostream>
#include <math.h>
#include <omp.h>
#define TILESIZE 16

using namespace std;
extern double deviceCall_time;
//extern double omptime;

//Dsub is data array
//Csub is centroid array
//Rsub is result array
//commonDin is the dimension of the data points
//Adim is the number  of the data points
//Bdimension number of cluster centers.
//dim_title is the tilesize of dimension array
void cudasafe( hipError_t error, char* message);
__global__ void mul(double *Dsub,double *Csub,double *Rsub,int commondim,int Adim,int Bdim,int dim_tile)
{
	 __shared__ double A[TILESIZE][TILESIZE];	//Declare shared memory array
	 __shared__ double B[TILESIZE][TILESIZE];

	double Cval=0;

	int gidx=blockDim.x*blockIdx.x + threadIdx.x;   //global index in x dimension
	int gidy=blockDim.y*blockIdx.y + threadIdx.y;   //global index in y dimension

	int rindex= gridDim.x*blockDim.x*gidy + gidx;   //global index for Rsub

	int dindex,cindex;             
	

	for(int m=0;m<((int)ceil( ((double)commondim)/TILESIZE ) );++m)
	{

		 dindex=blockIdx.y*commondim*blockDim.y + blockDim.x*m;	//IF d<16 then m always 0 ,hence blockDim.x*m is always 0. If d>=16 then it means d is multiple of 16 and thus blockDim.x=16
		 cindex=m*Bdim*blockDim.x + blockDim.y*blockIdx.x;

		 if(threadIdx.x < dim_tile)                             //limited by D
			A[threadIdx.y][threadIdx.x]=Dsub[dindex + threadIdx.y*commondim + threadIdx.x];	
	
		if(threadIdx.y < dim_tile)								//limited by D
			B[threadIdx.y][threadIdx.x]=Csub[cindex + threadIdx.y*Bdim + threadIdx.x];

		__syncthreads();
		
		for(int i=0;i<dim_tile;++i)
		{
			Cval+=(A[threadIdx.y][i]-B[i][threadIdx.x])*(A[threadIdx.y][i]-B[i][threadIdx.x]);
		}

		__syncthreads();

		//if(blockIdx.x==0 && blockIdx.y==0 )
			

	}
//	if(blockIdx.x==0 && blockIdx.y==127)
//	printf(" %f %d %d %d\n",Cval,row*Bdim+col,blockIdx.x,blockIdx.y);
	//printf("%d\n",rindex);
	Rsub[rindex]=sqrt(Cval);


}

int call_mul(double *h_data,double *h_centroid,double *h_result,int commondim,int Adim,int Bdim)
{
	int dim_tile;
	
	int gridx=Bdim/16,gridy=Adim/16;
	double *d_data,*d_centroid,*d_result;
	//printf("\n hi");
	cudasafe(hipMalloc(&d_data,sizeof(double)*Adim*commondim),"Allocate mem for d_data in matrix mul");
	cudasafe(hipMalloc(&d_centroid,sizeof(double)*commondim*Bdim),"Allocate mem for d_centroid in matr mul");
	cudasafe(hipMalloc(&d_result,sizeof(double)*Adim*Bdim),"Allocate mem for d_result in mat mul");

	hipMemcpy(d_data,h_data,sizeof(double)*Adim*commondim,hipMemcpyHostToDevice);
	hipMemcpy(d_centroid,h_centroid,sizeof(double)*commondim*Bdim,hipMemcpyHostToDevice);
	hipMemcpy(d_result,h_result,sizeof(double)*Adim*Bdim,hipMemcpyHostToDevice);


	dim3 grid(gridx,gridy);
	dim3 block(TILESIZE,TILESIZE);
	if(commondim < 16)
	{
		//size=sizeof(double)*TILESIZE*commondim;
		dim_tile=commondim;
	}
	else
	{
		//size=sizeof(double)*TILESIZE*TILESIZE;
		dim_tile=TILESIZE;
	}
	//timer.Start();
	clock_t begin = clock();
	//omptime=omp_get_wtime();
	mul<<<grid,block>>>(d_data,d_centroid,d_result,commondim,Adim,Bdim,dim_tile);	
	//timer.Stop();
	hipDeviceSynchronize();
	//omptime=omp_get_wtime()-omptime;
	//printf("Matrix Multiplication time %lf\n",omptime);
	clock_t end = clock();
	deviceCall_time += ((double)(end - begin)/CLOCKS_PER_SEC);
	
	hipMemcpy(h_result,d_result,sizeof(double)*Adim*Bdim,hipMemcpyDeviceToHost);

	
	//printf("\n Time=%g",timer.Elapsed());
	hipFree(d_data);
	hipFree(d_centroid);
	hipFree(d_result);
	return 0;
}
