//Creating 4 streams, each assigns a local thread index to the array
//

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 16
#define NCHUNK 2

//__device__ int *data;
__global__ 
void thread_multi(int t1,int *data)
{
        int i=blockDim.x * blockIdx.x + threadIdx.x;
        int j=threadIdx.x;
        printf(" %d %d\n",data[t1+i],threadIdx.x);
}

int main()
{
        int i=0;
        int *data;      
        hipStream_t stream[NCHUNK];
        size_t size = N*NCHUNK*sizeof(int);

       // cudaMalloc((void **)&d_t1, size);
       // cudaMallocHost(&h_t1, size);
        
        int *h_data=(int *)malloc(sizeof(int)*32);
        for(i=0;i<32;i++)
                h_data[i]=i;
        
        hipMalloc(&data,sizeof(int)*32);
        hipMemcpy(data,h_data,sizeof(int)*32,hipMemcpyHostToDevice);


        //for (i=0;i<N*NCHUNK;i++) {
          //      h_t1[i]=0;
        //}

//Create 4 streams
        for (i = 0; i < NCHUNK;i++) {
                hipStreamCreate(&stream[i]);
	}

//4 events on each stream - Memory copy to the device, execution, memory copy to the host, stream destroyed
    //    for(i=0;i<NCHUNK;i++) {
      //          cudaMemcpyAsync(d_t1+i*N, h_t1+i*N, N*sizeof(int), cudaMemcpyHostToDevice, stream[i]);
//	}
 //       for(i=0;i<NCHUNK;i++) {
   //             cudaStreamSynchronize(stream[i]);
     //   }

        for(i=0;i<NCHUNK;i++) {
                thread_multi<<<1,16,0,stream[i]>>>(i*16,data);
	}
        
        for(i=0;i<NCHUNK;i++) {
                hipStreamSynchronize(stream[i]);
        }

      /*  for(i=0;i<NCHUNK;i++) {
                cudaMemcpyAsync(h_t1+i*N, d_t1+i*N, N*sizeof(int), cudaMemcpyDeviceToHost, stream[i]);
	}

        for(i=0;i<NCHUNK;i++) {
                cudaStreamSynchronize(stream[i]);
        }*/
        for (i=0; i < NCHUNK; i++) {
                hipStreamDestroy(stream[i]);
	}
        
//Print result
       /* for(i=0;i<N*NCHUNK;i++) {
                printf("%d: %d\n",i, h_t1[i]);
        }*/       
       // cudaFree(d_t1);
        hipFree(data);
        printf("\nDone\n");
        return 0;
}
