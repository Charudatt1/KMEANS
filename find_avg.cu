
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <omp.h>
#define DEBUG
#define GRID_SIZE 1024

using namespace std;
extern double deviceCall_time;
void cudasafe( hipError_t error, char* message);
//extern double omptime;
//To divide sum of dimensions of documents belonging to same cluster by number of documents per cluster
__global__ void findAvgDevice(double *d_centroid,unsigned long long *d_unique,int Bdim,int Adim,int dim)
{

	int i;
	unsigned long long val;
	int id=blockDim.x*blockIdx.x+threadIdx.x;
	//TODO try to avoid thread divergence
	if(id+1 < Bdim)
		val =(d_unique[id+1]-d_unique[id]);
	else
		val =((Adim)-d_unique[id]);
	
	for(i=0;i<dim;i++)
	{
		d_centroid[i*Bdim+id]/=val;
	}
}

double *findAvg(double *h_centroid,unsigned long long *h_unique,int Adim,int Bdim,int dim)
{

	double *d_centroid;
	unsigned long long *d_unique;
	double *new_centroids=(double *)malloc(sizeof(double)*Bdim*dim);
	
	cudasafe(hipMalloc(&d_centroid,sizeof(double)*Bdim*dim),"Error findAvg d_centroid");
	cudasafe(hipMalloc(&d_unique,sizeof(unsigned long long)*Bdim),"Error findAvg d_unique");

	hipMemcpy(d_unique,h_unique,sizeof(unsigned long long)*Bdim, hipMemcpyHostToDevice);
	hipMemcpy(d_centroid,h_centroid,sizeof(double)*Bdim*dim, hipMemcpyHostToDevice);

		/*

	    for(int i=0;i<(Bdim);i++) 
	    {
			cout<<h_unique[i]<<" ";	    	
	    }
	    cout<<endl;
	    */
	int grid=(int)ceil((double)Bdim/1024);
	int block=min(1024,Bdim);
   	//block-1 to avoid thread divergence in kernel
	clock_t begin = clock();
	//omptime=omp_get_wtime();
   	findAvgDevice<<<grid,block>>>(d_centroid,d_unique,Bdim,Adim,dim);  
   	hipDeviceSynchronize();
   	//omptime=omp_get_wtime()-omptime;
   	clock_t end = clock();
	//printf("Find average time %lf\n",omptime);
	deviceCall_time += ((double)(end - begin)/CLOCKS_PER_SEC);
   	hipMemcpy(new_centroids,d_centroid,sizeof(double)*Bdim*dim, hipMemcpyDeviceToHost);

/*
   for(int i=0;i<(Bdim);i++) 
	    {
	    	
	    	for (int j = 0; j < dim; j++)
	    	{
	    		cout<<new_centroids[j*Bdim+i]<<" ";

	    	}
	    	cout<<endl;
	    	      
	    }
	    */

	hipFree(d_centroid);
	hipFree(d_unique);

   	return new_centroids;
}