#include "hip/hip_runtime.h"
#include <stdio.h>
#include <float.h>
#include <fstream>
#include "gputimer.h"
#include <bitset>
#include <omp.h>

//#define DEBUG1

using namespace std;
extern double deviceCall_time;
//extern double omptime;

void cudasafe( hipError_t error, char* message);

//minimization method takes in the Input array whose dimensions are K*N(i.e centroids * data_points)
//minCentroid is an array of size data_points . Each index in the minCentroid points to one value between 0- (centroids-1)

__global__ void minimization(double *d_Input,int centroids,int data_points,unsigned long long *d_minCentroid)
{
	double minimum=FLT_MAX;               //Variable to store the Minimum value initialized with max value of double
	unsigned int minIndex=0;						 //Variable to contain the index of the minimum value
	double val;							 //variable to store the value from the array

	unsigned long long gid=blockIdx.x*blockDim.x + threadIdx.x;

	for(int i=0;i<centroids;i++)
	{
		
		if( gid < data_points)
			val = d_Input[i*data_points + gid];
		else
			val = FLT_MAX;
	
	//	int z= ( data_points - 1 - (int)gid)>>31;       ///performance improvement achieved by removing if statement while reading
   // 	val = (1-z)*d_Input[i*data_points + (int)gid] + z*FLT_MAX;

		if(val < minimum)
		{
			minimum = val;
			minIndex = i;
		}
	}

	if((int)gid < data_points )
		d_minCentroid[gid] = gid<<32 | minIndex;


}


 unsigned long long* compute_minimization(double* d_Input,int data_points,int centroids)
 {
 	unsigned long long *d_minCentroid,*h_minCentroid;     //declare device and global memories
 
 	h_minCentroid=(unsigned long long*)malloc(sizeof(unsigned long long)*data_points);   //declare global centroid

 	cudasafe(hipMalloc(&d_minCentroid,sizeof(unsigned long long)*data_points),"allocating memory in d_minCentroid");    //allocate memory for the device
  
 	dim3 block(1024);                                      //global block threads size is 1024
 	dim3 grid((int)(ceil((double)data_points/1024)));	   //grid size is datapoints/1024

 	clock_t begin = clock();
 	//omptime=omp_get_wtime();
 	minimization<<<block,grid>>>(d_Input,centroids,data_points,d_minCentroid); //call to function
 	clock_t end = clock();
 	//omptime=omp_get_wtime()-omptime;
 	//printf("Minimization time %lf\n",omptime);
 	
	deviceCall_time += ((double)(end - begin)/CLOCKS_PER_SEC);

 	cudasafe(hipMemcpy(h_minCentroid,d_minCentroid,sizeof(unsigned long long)*data_points,hipMemcpyDeviceToHost),"Copying from d_minCentroid to h_minCentroid");

 	cudasafe(hipFree(d_minCentroid),"freeing memory");
 	cudasafe(hipFree(d_Input),"Free d_input function name 'compute_minimization' line 70");

 	#ifdef DEBUG1

		fstream fout;

		fout.open("minimumcentroid.txt",ios::out);
		//printf("\n");

		//4294967295
		unsigned long long offset=4294967295;

		for(int i=0;i<data_points;i++)
		{
																						//std::bitset<64> x(h_minCentroid[i]);
			
			fout << ((h_minCentroid[i]) & offset) << endl; //print into a file
																						//fout << x << endl; 
		
			//printf("%llu \n",((h_minCentroid[i]) & (unsigned long long)4294967295));
		}	

		fout<<endl;

		fout.close();

	#endif


 	return h_minCentroid;
 }
